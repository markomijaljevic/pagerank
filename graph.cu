#include "hip/hip_runtime.h"
#include "Graph.h"
#include <string>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""


void Graph::addNewNode(Node* newNode) {
	newNode->sizeOfOutLinks = newNode->getOutGoingLinks().size();
	this->Nodes.push_back(newNode);
}

size_t Graph::getNodeCount() {
	return this->Nodes.size();
}


Node* Graph::getInLinksNodes(thrust::host_vector<Node*> links) {

	this->inLinks = new Node[links.size()];
	int i = 0;
	for (Node* node_ptr : links) {
		inLinks[i] = *node_ptr;
		i++;
	}
	return &inLinks[0];
}

void Graph::printGraphInfo() {
	for (Node* node : this->Nodes) {
		std::cout << "Url: " << node->getUrl() << std::endl;
		std::cout << "PR: " << (node->getCurrentPR() / this->normalize_sum ) << std::endl;
		std::cout << "In Going Links: " << node->getInGoingLinks().size() << std::endl;
		std::cout << "Out Going Links: " << node->getOutGoingLinks().size() << std::endl;
		std::cout << std::endl;
	}
	std::cout << "Number of iterations: " << this->iter_counter << std::endl;
	std::cout << std::endl;
}


__global__ void calculatePR( Node* dev_inLinks, float* dev_PR) {

	int i = threadIdx.x;
	dev_PR[i] = ( dev_inLinks[i].PRinLastIter / dev_inLinks[i].sizeOfOutLinks );
}

__global__ void updateLastPR( Node* dev_Nodes, bool* flag, float* dev_sum) {

	int i = threadIdx.x;
	if (dev_Nodes[i].PRinLastIter != dev_Nodes[i].PRinCurrentIter)
		*flag = true;

	dev_Nodes[i].PRinLastIter = dev_Nodes[i].PRinCurrentIter;
	dev_sum[i] = dev_Nodes[i].PRinCurrentIter;
}


void Graph::PageRank_GPU() {
	
	int sizeOfInLinks;
	Node* inLinks;
	Node* dev_inLinks;
	Node* dev_Nodes;
	Node* temp_Nodes;
	bool flag = true;
	bool* dev_flag;
	float* dev_normalize_sum;
	float* dev_PR;
	float PR;

	hipError_t cudaStatus;


	while (flag) {

		flag = false;
		this->normalize_sum = 0;

		for (Node* node : this->Nodes) {

			sizeOfInLinks = node->getInGoingLinks().size();

			if (sizeOfInLinks > 0) {

				inLinks = getInLinksNodes(node->inlinks);

				cudaStatus = hipSetDevice(0);
				if (cudaStatus != hipSuccess) {
					std::cout << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?" << std::endl;
					return;
				}

				cudaStatus = hipMalloc((void**)&dev_inLinks, sizeOfInLinks * sizeof(Node));
				if (cudaStatus != hipSuccess) {
					std::cout << "hipMalloc failed" << std::endl;
					return;
				}

				cudaStatus = hipMalloc((void**)&dev_PR, sizeOfInLinks * sizeof(float));
				if (cudaStatus != hipSuccess) {
					std::cout << "hipMalloc failed" << std::endl;
					return;
				}

				cudaStatus = hipMemcpy(dev_inLinks, inLinks, sizeOfInLinks * sizeof(Node), hipMemcpyHostToDevice);
				if (cudaStatus != hipSuccess) {
					std::cout << "hipMemcpy failed" << std::endl;
					return;
				}

				calculatePR << < 1, sizeOfInLinks >> > (dev_inLinks, dev_PR);

				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					std::cout << "calculatePR launch failed: " <<  hipGetErrorString(cudaStatus) << std::endl;;
					return;
				}

				cudaStatus = hipDeviceSynchronize();
				if (cudaStatus != hipSuccess) {
					std::cout << "hipDeviceSynchronize returned error code " << cudaStatus << "after launching calculatePR!" << std::endl;
					 return;
				}

				//wrap raw pointer with a device_ptr
				thrust::device_ptr<float> dev_ptr_PR(dev_PR);
				//copy memory to a new dev vector , and use vector 
				thrust::device_vector<float> PageRankValue(dev_ptr_PR, dev_ptr_PR + sizeOfInLinks);

				//thrust::copy(PageRankValue.begin(), PageRankValue.end(), std::ostream_iterator<float>(std::cout, "\n"));
				PR = thrust::reduce(PageRankValue.begin(), PageRankValue.end(), (float)0, thrust::plus<float>());
				/*PageRankValue.clear();
				PageRankValue.shrink_to_fit();*/

				delete[] this->inLinks;
				this->inLinks = 0;
				PageRankValue.clear();
				PageRankValue.shrink_to_fit();
			}

			PR = (1 - d) + (d * PR);
			node->setPR(PR);
			PR = 0;

			if (sizeOfInLinks > 0) {
				hipFree(dev_inLinks);
				hipFree(dev_PR);
				dev_PR = 0;
				dev_inLinks = 0;
				inLinks = 0;
			}

		}//end of for

		temp_Nodes = getInLinksNodes(this->Nodes);
	
		sizeOfInLinks = this->Nodes.size();

		cudaStatus = hipMalloc((void**)&dev_Nodes, sizeOfInLinks * sizeof(Node));
		if (cudaStatus != hipSuccess) {
			std::cout << "hipMalloc failed" << std::endl;
			return;
		}

		cudaStatus = hipMalloc((void**)&dev_flag, sizeof(bool));
		if (cudaStatus != hipSuccess) {
			std::cout << "hipMalloc failed" << std::endl;
			return;
		}

		cudaStatus = hipMalloc((void**)&dev_normalize_sum, sizeOfInLinks * sizeof(float));
		if (cudaStatus != hipSuccess) {
			std::cout << "hipMalloc failed" << std::endl;
			return;
		}


		cudaStatus = hipMemcpy(dev_Nodes, temp_Nodes, sizeOfInLinks * sizeof(Node), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			std::cout << "hipMemcpy failed" << std::endl;
			return;
		}

		updateLastPR << <1, sizeOfInLinks >> > (dev_Nodes, dev_flag, dev_normalize_sum);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			std::cout << "updateLastPR launch failed: " << hipGetErrorString(cudaStatus) << std::endl;;
			return;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			std::cout << "hipDeviceSynchronize returned error code " << cudaStatus << "after launching updateLastPR!" << std::endl;
			return;
		}

		cudaStatus = hipMemcpy(temp_Nodes, dev_Nodes, sizeOfInLinks * sizeof(Node), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			std::cout << "hipMemcpy failed" << std::endl;
			return;
		}

		cudaStatus = hipMemcpy(&flag, dev_flag, sizeof(bool), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			std::cout << "hipMemcpy failed" << std::endl;
			return;
		}

		thrust::device_ptr<float> dev_ptr_sum(dev_normalize_sum);
		thrust::device_vector<float> norm_sum(dev_ptr_sum, dev_ptr_sum + sizeOfInLinks);

		this->normalize_sum = thrust::reduce(norm_sum.begin(), norm_sum.end(), (float)0, thrust::plus<float>());

		hipFree(dev_Nodes);
		hipFree(dev_flag);
		hipFree(dev_normalize_sum);
		delete[] this->inLinks;
		this->inLinks = 0;

		norm_sum.clear();
		norm_sum.shrink_to_fit();

		this->iter_counter++;

	}//end of while


	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		std::cout << "hipDeviceReset failed!" << std::endl;
		return;
	}


}//end of for


